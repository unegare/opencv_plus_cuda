#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include "libfilt_001.h"

__global__ void kernel (uchar *arr, int rows, int cols) {
	int tid = (threadIdx.x + blockIdx.x * blockDim.x)*3;
	int step = gridDim.x * blockDim.x * 3;
	int N = rows * cols * 3;
	while (tid < N) {
		arr[tid] = 255;//(arr[tid] + arr[tid +1] + arr[tid +2])/3;
//		if ((tid%rows)%20 < 10) {
//			arr[tid] = 255;
//			arr[tid +1] = 0;
//			arr[tid +2] = 0;
//		} else {
//			arr[tid] = 255;
//			arr[tid +1] = 255;
//			arr[tid +2] = 255;
//		}
		tid += step;
	}
}

namespace _filt {
	_Dev_ptr::_Dev_ptr() : _dev_arr(NULL) {}
	_Dev_ptr::~_Dev_ptr() {
		if (_dev_arr) {
			printf ("_filt::_dev_ptr._dev_arr == %p\n", _dev_arr);
			hipError_t err = hipFree ( _dev_arr );
			if ( err != hipSuccess ) {
				std::cout << "ERROR: hipFree ( _filt::_dev_ptr._dev_arr ) : " << err << " : " << hipGetErrorString(err) << std::endl;
			}
//			std::cout << "LOG 3" << std::endl;
		}
	}
	_Dev_ptr::operator uchar* () const {
		return _dev_arr;
	}
	_Dev_ptr::operator bool () const {
		return _dev_arr;
	}
	uchar*& _Dev_ptr::operator* () {
		return _dev_arr;
	}
	uchar* _Dev_ptr::operator* () const {
		return _dev_arr;
	}
	//_Dev_ptr _dev_ptr; // udefined behaviour in the destructor -> hipError_t 29 "driver shutting down"
}

void filt (cv::Mat &m, _filt::_Dev_ptr *_dev_arr) {
//	static bool print = true;
	uchar *dev_arr = NULL;
	int rows(m.rows);
	int cols(m.cols);
	if (_dev_arr) {
		dev_arr = **_dev_arr;
	}
	if (dev_arr == NULL) {
		if ( hipMalloc ( (void**)&dev_arr, rows*cols*3*sizeof(uchar) ) != hipSuccess ) {
			std::cout << "ERROR: hipMalloc error" << std::endl;
			return;
		}
//		std::cout << "LOG 1" << std::endl;
		if (_dev_arr) {
			**_dev_arr = dev_arr;
		}
	}
  switch (m.type()) {
    case CV_8UC3: {
//			if (print) {
//				std::cout << "CV_8UC3" << std::endl;
//				print = false;
//			}
			if ( hipMemcpy ( dev_arr, m.data, rows*cols*3*sizeof(uchar), hipMemcpyHostToDevice ) != hipSuccess ) {
				std::cout << "ERROR: hipMemcpyHostToDevice" << std::endl;
				break;
			}
			kernel<<<32,32>>> ( dev_arr, rows, cols );
			if ( hipMemcpy ( m.data, dev_arr, rows*cols*3*sizeof(uchar), hipMemcpyDeviceToHost ) != hipSuccess ) {
				std::cout << "ERROR: hipMemcpyDeviceToHost" << std::endl;
				break;
			}
      break;
    }
    default: {
      std::cout << "m.type() == " << m.type() << " is unknown" << std::endl;
    }
  }

	if (_dev_arr) {
	} else {
		if ( hipFree ( dev_arr ) != hipSuccess ) {
			std::cout << "ERROR: hipFree" << std::endl;
		}
//		std::cout << "LOG 2" << std::endl;
	}
}

